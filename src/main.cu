#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cmath>
#include <complex>
#include <iostream>
#include <fstream>
#include <new>
#include <cmath>
#include <chrono>

// The NVIDIA CUDA cuFFT library
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <>

using namespace std;
using namespace chrono;

constexpr size_t N		(1024);
constexpr size_t B		(1);


__global__ void bpfilter(hipfftComplex* c)
{
	int i = threadIdx.x;
	if (i != 30)
	{
		c[i].x = 0.0;
		c[i].y = 0.0;
	}
}
__global__ void multiScalar(hipfftComplex* c, const hipfftComplex* a, const float b)
{
	int i = threadIdx.x;
	c[i].x = a[i].x * b;
	c[i].y = a[i].y * b;
}

int main(int argc, char* const __restrict argv[])
{
	ofstream outputFile("filtered.csv");
	ifstream inputFile("data.txt");
	hipfftComplex* pf8Signal_device(nullptr);
	complex<float>* pf8Signal_host(new(nothrow) complex<float>[N * B]);
	float f8Intermediate(0.0);
	hipfftHandle plan;
	hipfftResult status;

	for (unsigned int idx = 0; idx < (N * B); idx++)
	{
		inputFile >> f8Intermediate;
		pf8Signal_host[idx].real(f8Intermediate);
		pf8Signal_host[idx].imag(0.0);
	}
	inputFile.close();

	hipMalloc(&pf8Signal_device, sizeof(hipfftComplex) * N * B);
	hipMemcpy(pf8Signal_device, pf8Signal_host, sizeof(hipfftComplex) * N * B, hipMemcpyHostToDevice);
	status = hipfftPlan1d(&plan, N, HIPFFT_C2C, B);

	status = hipfftExecC2C(plan, pf8Signal_device, pf8Signal_device, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	bpfilter<<<B, N>>>(pf8Signal_device);
	status = hipfftExecC2C(plan, pf8Signal_device, pf8Signal_device, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	multiScalar<<<B, N>>>(pf8Signal_device, pf8Signal_device, 1.0F / (N * B));

	hipMemcpy(pf8Signal_host, pf8Signal_device, sizeof(hipfftComplex) * N * B, hipMemcpyDeviceToHost);
	for (unsigned int idx = 0; idx < (N * B); idx++)
	{
		outputFile << pf8Signal_host[idx].real() * 10.0f << ',' << endl;
	}
	outputFile.close();

	hipfftDestroy(plan);
	hipFree(pf8Signal_device);
	delete[] pf8Signal_host;
	return NULL;
}